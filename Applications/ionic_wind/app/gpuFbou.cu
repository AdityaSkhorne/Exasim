
#include <hip/hip_runtime.h>
template <typename T>  __device__  void devicegpuFbou1(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T tau1 = tau[0];
		T xdg1 = xdg[0*ng+i];
		T udg2 = udg[1*ng+i];
		T udg4 = udg[3*ng+i];
		T udg6 = udg[5*ng+i];
		T uhg2 = uhg[1*ng+i];
		T odg2 = odg[1*ng+i];
		T odg3 = odg[2*ng+i];
		T nlg1 = nlg[0*ng+i];
		T nlg2 = nlg[1*ng+i];
		f[0*ng+i] = 0.0;
		f[1*ng+i] = tau1*(udg2-uhg2)+nlg1*xdg1*(odg2+udg4)+nlg2*xdg1*(odg3+udg6);
		i += blockDim.x * gridDim.x;
	}
}

template <typename T>  __global__  void kernelgpuFbou1(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFbou1(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T>  __device__  void devicegpuFbou2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		f[0*ng+i] = 0.0;
		f[1*ng+i] = 0.0;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T>  __global__  void kernelgpuFbou2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFbou2(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T>  __device__  void devicegpuFbou3(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T tau1 = tau[0];
		T xdg1 = xdg[0*ng+i];
		T udg1 = udg[0*ng+i];
		T udg2 = udg[1*ng+i];
		T udg3 = udg[2*ng+i];
		T udg4 = udg[3*ng+i];
		T udg5 = udg[4*ng+i];
		T udg6 = udg[5*ng+i];
		T uhg1 = uhg[0*ng+i];
		T uhg2 = uhg[1*ng+i];
		T odg2 = odg[1*ng+i];
		T odg3 = odg[2*ng+i];
		T nlg1 = nlg[0*ng+i];
		T nlg2 = nlg[1*ng+i];
		T t2 = odg2+udg4;
		T t3 = odg3+udg6;
		T t4 = -uhg1;
		T t5 = t4+udg1;
		T t7 = nlg1*t2*1.0E+3;
		T t8 = nlg2*t3*1.0E+3;
		T t6 = t5*tau1;
		T t9 = t7+t8;
		T t10 = tanh(t9);
		T t11 = t10/2.0;
		f[0*ng+i] = -(t6-udg1*xdg1*(nlg1*t2+nlg2*t3))*(t11-1.0/2.0)+(t11+1.0/2.0)*(t6+nlg1*xdg1*(udg3/1.0E+1-t2*udg1)+nlg2*xdg1*(udg5/1.0E+1-t3*udg1));
		f[1*ng+i] = tau1*(udg2-uhg2)+nlg1*t2*xdg1+nlg2*t3*xdg1;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T>  __global__  void kernelgpuFbou3(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFbou3(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T>  __device__  void devicegpuFbou4(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T tau1 = tau[0];
		T xdg1 = xdg[0*ng+i];
		T udg1 = udg[0*ng+i];
		T udg2 = udg[1*ng+i];
		T udg4 = udg[3*ng+i];
		T udg6 = udg[5*ng+i];
		T uhg1 = uhg[0*ng+i];
		T uhg2 = uhg[1*ng+i];
		T odg2 = odg[1*ng+i];
		T odg3 = odg[2*ng+i];
		T nlg1 = nlg[0*ng+i];
		T nlg2 = nlg[1*ng+i];
		T t2 = odg2+udg4;
		T t3 = odg3+udg6;
		f[0*ng+i] = tau1*(udg1-uhg1)-udg1*xdg1*(nlg1*t2+nlg2*t3);
		f[1*ng+i] = tau1*(udg2-uhg2)+nlg1*t2*xdg1+nlg2*t3*xdg1;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T>  __global__  void kernelgpuFbou4(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFbou4(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T>  __device__  void devicegpuFbou5(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		f[0*ng+i] = 0.0;
		f[1*ng+i] = 0.0;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T>  __global__  void kernelgpuFbou5(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFbou5(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T> void gpuFbou(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	if (ib == 1)
		kernelgpuFbou1<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 2)
		kernelgpuFbou2<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 3)
		kernelgpuFbou3<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 4)
		kernelgpuFbou4<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 5)
		kernelgpuFbou5<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuFbou(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int);
template void gpuFbou(float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int);
#ifdef _ENZYME
template <typename T> __global__ void kernelGradgpuFbou1Enzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *dodg, T *wdg, T *dwdg, T *uhg, T *duhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	__enzyme_fwddiffgpuFbou((void*)devicegpuFbou1<T>,
			  enzyme_dup, f, df,
			 enzyme_const, xg,
			 enzyme_dup, udg, dudg,
			 enzyme_dup, odg, dodg,
			 enzyme_dup, wdg, dwdg,
			 enzyme_dup, uhg, duhg,
			 enzyme_const, nlg,
			 enzyme_const, tau,
			 enzyme_const, uinf,
			 enzyme_const, param,
			 enzyme_const, time,
			 enzyme_const, modelnumber,
			 enzyme_const, ng,
			 enzyme_const, nc,
			 enzyme_const, ncu,
			 enzyme_const, nd,
			 enzyme_const, ncx,
			 enzyme_const, nco,
			 enzyme_const, ncw);
}

template <typename T> __global__ void kernelGradgpuFbou2Enzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *dodg, T *wdg, T *dwdg, T *uhg, T *duhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	__enzyme_fwddiffgpuFbou((void*)devicegpuFbou2<T>,
			  enzyme_dup, f, df,
			 enzyme_const, xg,
			 enzyme_dup, udg, dudg,
			 enzyme_dup, odg, dodg,
			 enzyme_dup, wdg, dwdg,
			 enzyme_dup, uhg, duhg,
			 enzyme_const, nlg,
			 enzyme_const, tau,
			 enzyme_const, uinf,
			 enzyme_const, param,
			 enzyme_const, time,
			 enzyme_const, modelnumber,
			 enzyme_const, ng,
			 enzyme_const, nc,
			 enzyme_const, ncu,
			 enzyme_const, nd,
			 enzyme_const, ncx,
			 enzyme_const, nco,
			 enzyme_const, ncw);
}

template <typename T> __global__ void kernelGradgpuFbou3Enzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *dodg, T *wdg, T *dwdg, T *uhg, T *duhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	__enzyme_fwddiffgpuFbou((void*)devicegpuFbou3<T>,
			  enzyme_dup, f, df,
			 enzyme_const, xg,
			 enzyme_dup, udg, dudg,
			 enzyme_dup, odg, dodg,
			 enzyme_dup, wdg, dwdg,
			 enzyme_dup, uhg, duhg,
			 enzyme_const, nlg,
			 enzyme_const, tau,
			 enzyme_const, uinf,
			 enzyme_const, param,
			 enzyme_const, time,
			 enzyme_const, modelnumber,
			 enzyme_const, ng,
			 enzyme_const, nc,
			 enzyme_const, ncu,
			 enzyme_const, nd,
			 enzyme_const, ncx,
			 enzyme_const, nco,
			 enzyme_const, ncw);
}

template <typename T> __global__ void kernelGradgpuFbou4Enzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *dodg, T *wdg, T *dwdg, T *uhg, T *duhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	__enzyme_fwddiffgpuFbou((void*)devicegpuFbou4<T>,
			  enzyme_dup, f, df,
			 enzyme_const, xg,
			 enzyme_dup, udg, dudg,
			 enzyme_dup, odg, dodg,
			 enzyme_dup, wdg, dwdg,
			 enzyme_dup, uhg, duhg,
			 enzyme_const, nlg,
			 enzyme_const, tau,
			 enzyme_const, uinf,
			 enzyme_const, param,
			 enzyme_const, time,
			 enzyme_const, modelnumber,
			 enzyme_const, ng,
			 enzyme_const, nc,
			 enzyme_const, ncu,
			 enzyme_const, nd,
			 enzyme_const, ncx,
			 enzyme_const, nco,
			 enzyme_const, ncw);
}

template <typename T> __global__ void kernelGradgpuFbou5Enzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *dodg, T *wdg, T *dwdg, T *uhg, T *duhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	__enzyme_fwddiffgpuFbou((void*)devicegpuFbou5<T>,
			  enzyme_dup, f, df,
			 enzyme_const, xg,
			 enzyme_dup, udg, dudg,
			 enzyme_dup, odg, dodg,
			 enzyme_dup, wdg, dwdg,
			 enzyme_dup, uhg, duhg,
			 enzyme_const, nlg,
			 enzyme_const, tau,
			 enzyme_const, uinf,
			 enzyme_const, param,
			 enzyme_const, time,
			 enzyme_const, modelnumber,
			 enzyme_const, ng,
			 enzyme_const, nc,
			 enzyme_const, ncu,
			 enzyme_const, nd,
			 enzyme_const, ncx,
			 enzyme_const, nco,
			 enzyme_const, ncw);
}

template <typename T> void gpuFbouEnzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *dodg, T *wdg, T *dwdg, T *uhg, T *duhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	if (ib == 1)
		kernelGradgpuFbou1Enzyme<<<gridDim, blockDim>>>(f, df, xg, udg, dudg, odg, dodg, wdg, dwdg, uhg, duhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 2)
		kernelGradgpuFbou2Enzyme<<<gridDim, blockDim>>>(f, df, xg, udg, dudg, odg, dodg, wdg, dwdg, uhg, duhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 3)
		kernelGradgpuFbou3Enzyme<<<gridDim, blockDim>>>(f, df, xg, udg, dudg, odg, dodg, wdg, dwdg, uhg, duhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 4)
		kernelGradgpuFbou4Enzyme<<<gridDim, blockDim>>>(f, df, xg, udg, dudg, odg, dodg, wdg, dwdg, uhg, duhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (ib == 5)
		kernelGradgpuFbou5Enzyme<<<gridDim, blockDim>>>(f, df, xg, udg, dudg, odg, dodg, wdg, dwdg, uhg, duhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuFbouEnzyme(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int);
#endif