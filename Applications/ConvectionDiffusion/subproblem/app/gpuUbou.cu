#include "gpuUbou1.cu"
#include "gpuUbou2.cu"

template <typename T> void gpuUbou(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	if (modelnumber == 1)
		gpuUbou1(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (modelnumber == 2)
		gpuUbou2(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuUbou(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int);
template void gpuUbou(float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int);
