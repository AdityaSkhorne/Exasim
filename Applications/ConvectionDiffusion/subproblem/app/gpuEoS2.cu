
#include <hip/hip_runtime.h>
template <typename T> void gpuEoS2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

template void gpuEoS2(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int, int, int);
template void gpuEoS2(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int, int, int);

template <typename T> void __device__ devicegpuEoS2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

