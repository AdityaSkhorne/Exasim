
#include <hip/hip_runtime.h>
template <typename T> void gpuEoSdw2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

template void gpuEoSdw2(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int, int, int);
template void gpuEoSdw2(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int, int, int);

template <typename T> void __device__ devicegpuEoSdw2(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

