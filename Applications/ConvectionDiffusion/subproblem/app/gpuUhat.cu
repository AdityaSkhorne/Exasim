#include "gpuUhat1.cu"
#include "gpuUhat2.cu"

template <typename T> void gpuUhat(T *f, T *xdg, T *udg1, T *udg2,  T *odg1, T *odg2,  T *wdg1, T *wdg2,  T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	if (modelnumber == 1)
		gpuUhat1(f, xdg, udg1, udg2, odg1, odg2, wdg1, wdg2, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
	else if (modelnumber == 2)
		gpuUhat2(f, xdg, udg1, udg2, odg1, odg2, wdg1, wdg2, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuUhat(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double time, int, int, int, int, int, int, int, int);
template void gpuUhat(float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float time, int, int, int, int, int, int, int, int);
