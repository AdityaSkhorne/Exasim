
#include <hip/hip_runtime.h>
template <typename T>  __device__  void devicegpuEoSdu(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		int j = i%npe;
		int k = (i-j)/npe;
		T xdg1 = xdg[j+npe*0+npe*ncx*k];
		T xdg2 = xdg[j+npe*1+npe*ncx*k];
		T udg1 = udg[j+npe*0+npe*nc*k];
		T udg2 = udg[j+npe*1+npe*nc*k];
		T udg3 = udg[j+npe*2+npe*nc*k];
		T wdg1 = wdg[j+npe*0+npe*ncw*k];
		f[j+npe*0+npe*nce*k] = udg1*-2.0;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> __global__ void kernelgpuEoSdu(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
	devicegpuEoSdu(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw, nce, npe, ne);
}

template <typename T> void gpuEoSdu(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuEoSdu<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw, nce, npe, ne);
}

template void gpuEoSdu(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int, int, int);
template void gpuEoSdu(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int, int, int);
